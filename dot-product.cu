
#include <hip/hip_runtime.h>
#include <cstddef>
#include <chrono>
#include <iostream>
#include <random>
#include <vector>

#define BLOCK_SIZE 256

__global__ void dot(const float *a, const float *b, float *result, int n)
{
    __shared__ float shared_mem[BLOCK_SIZE];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
        shared_mem[tid] = a[idx] * b[idx];
    __syncthreads();

    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s)
            shared_mem[tid] += shared_mem[tid + s];
        __syncthreads();
    }

    if (tid == 0)
        atomicAdd(result, shared_mem[0]);
}

int main()
{
    const std::size_t n = 500000;

    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937 engine(seed);
    const int width = 100;
    std::uniform_int_distribution random_value(-width, width);

    std::vector<float> a(n);
    for (float &value: a)
        value = random_value(engine) / 100.0;
    std::vector<float> b(n);
    for (float &value: b)
        value = random_value(engine) / 100.0;

    float *d_a, *d_b, *d_result;
    float result = 0.0;
    const std::size_t num_bytes = n * sizeof(float);
    hipMalloc(&d_a, num_bytes);
    hipMalloc(&d_b, num_bytes);
    hipMalloc(&d_result, sizeof(float));
    hipMemcpy(d_a, (const void *) a.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, (const void *) b.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start_gpu, end_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&end_gpu);

    // == (n/BLOCK_SIZE + (n%BLOCK_SIZE != 0))
    std::size_t gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEventRecord(start_gpu);
    dot<<<gridSize, BLOCK_SIZE>>>(d_a, d_b, d_result, n);

    hipEventRecord(end_gpu);
    hipEventSynchronize(end_gpu);
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    float time_gpu = 0;
    hipEventElapsedTime(&time_gpu, start_gpu, end_gpu);
    hipEventDestroy(start_gpu);
    hipEventDestroy(end_gpu);
    std::cerr << "GPU: " << "" << time_gpu << "ms, result = " << result << std::endl;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    result = 0.0;
    auto start_cpu = std::chrono::high_resolution_clock::now();
    for (std::size_t i = 0; i < n; ++i)
        result += a[i] * b[i];
    auto end_cpu = std::chrono::high_resolution_clock::now();
    const float time_cpu = std::chrono::duration_cast<std::chrono::microseconds>(end_cpu - start_cpu).count() / 1000.0f;
    std::cerr << "CPU: " << "" << time_cpu << "ms, result = " << result << std::endl;

    return 0;
}
