
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstddef>
#include <set>
#include <vector>
#include <random>
#include <chrono>

class CSR_Matrix
{
public:
    using size_t = std::size_t;
    using vector_d = std::vector<double>;
    using vector_i = std::vector<size_t>;

private:
    size_t m_num_rows;
    size_t m_num_columns;
    vector_d m_values;
    vector_i m_columns;
    vector_i m_indices;
public:
    struct Triple
    {
        size_t row;
        size_t column;
        double value;

        // TODO create default trash-filling constructor (?)

        Triple(const size_t row = 0, const size_t column = 0,
               const double value = 0.0)
            : row(row), column(column), value(value)
        {}

        class Compare {
        public:
            bool operator()(const Triple &a, const Triple &b) const
            {
                if (a.row < b.row)
                    return true;
                if (a.row == b.row && a.column < b.column)
                    return true;
                return false;
            }
        };
    };

    CSR_Matrix(const size_t num_rows, const size_t num_columns)
        : m_num_rows(num_rows), m_num_columns(num_columns)
    {}

    size_t num_rows() const { return m_num_rows; }
    size_t num_columns() const { return m_num_columns; }
    const vector_d values() const { return m_values; }
    const vector_i columns() const { return m_columns; }
    const vector_i indices()  const { return m_indices; };

    static CSR_Matrix random(const size_t num_rows, const size_t num_columns,
                             const size_t num_nonzero)
    {
        std::mt19937 engine(112233);
        std::uniform_int_distribution<size_t> random_row(0, num_rows-1);
        std::uniform_int_distribution<size_t> random_column(0, num_columns-1);
        const int width = 100;
        std::uniform_int_distribution random_value(-width, width);

        std::set<Triple, Triple::Compare> triples;
        for (size_t i = 0; i < num_nonzero; ++i) {
            triples.insert(Triple(
                random_row(engine),
                random_column(engine),
                static_cast<double>(random_value(engine)) / static_cast<double>(width)
                ));
        }

        CSR_Matrix matrix(num_rows, num_columns);
        matrix.m_indices.push_back(0); // padding index
        size_t row = 0;
        size_t i = 0;
        for (const Triple &triple: triples) {
            while (row < triple.row) {
                matrix.m_indices.push_back(i);
                ++row;
            }
            matrix.m_values.push_back(triple.value);
            matrix.m_columns.push_back(triple.column);
            ++i;
        }
        matrix.m_indices.push_back(i); // padding index

        return matrix;
    }

    friend std::ostream &operator<<(std::ostream &out, const CSR_Matrix &matrix)
    {
        size_t i = 1;
        for (; i < matrix.m_indices.size(); ++i) {
            size_t j = 0;
            for (size_t col = matrix.m_indices[i-1];
                 j < matrix.num_columns() && col < matrix.m_indices[i]; ++j)
            {
                if (j == matrix.m_columns[col]) {
                    out << matrix.m_values[col];
                    ++col;
                } else {
                    out << 0.0;
                }
                out << "\t";
            }
            for (; j < matrix.num_columns(); ++j)
                out << 0.0 << "\t";
            out << std::endl;
        }
        return out;
    }

    vector_d operator*(const vector_d &a) const
    {
        vector_d result(a.size()); // filled with zeros by default

        for (size_t i = 1, j = 0; i < m_indices.size(); ++i, ++j)
            for (size_t k = m_indices[i-1]; k < m_indices[i]; ++k)
                result[j] += m_values[k] * a[m_columns[k]];

        return result;
    }
};

__global__ void multiply(CSR_Matrix::size_t num_values,
                         CSR_Matrix::size_t num_indices,
                         double *values,
                         CSR_Matrix::size_t *columns,
                         CSR_Matrix::size_t *indices,
                         double *a,
                         double *result)
{
    CSR_Matrix::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_indices) {
        CSR_Matrix::size_t j = i - 1;
        result[j] = 0.0;
        for (size_t k = indices[i-1]; k < indices[i]; ++k)
            result[j] += values[k] * a[columns[k]];
    }
}

int main(void)
{
    const CSR_Matrix::size_t num_rows = 5000;
    const CSR_Matrix::size_t num_columns = 5000;
    const CSR_Matrix::size_t num_nonzero = 5000000;

    auto matrix = CSR_Matrix::random(num_rows, num_columns, num_nonzero);

    std::mt19937 engine(112233);
    const int width = 100;
    std::uniform_int_distribution random_value(-width, width);
    CSR_Matrix::vector_d a(num_columns);
    for (double &value: a)
        value = random_value(engine);

    double *d_values;
    const std::size_t bytes_values = matrix.values().size() * sizeof(double);
    CSR_Matrix::size_t *d_columns;
    const std::size_t bytes_columns = matrix.columns().size() * sizeof(CSR_Matrix::size_t);
    CSR_Matrix::size_t *d_indices;
    const std::size_t bytes_indices = matrix.indices().size() * sizeof(CSR_Matrix::size_t);
    double *d_a;
    const std::size_t bytes_a = a.size() * sizeof(double);
    double *d_result;
    const std::size_t bytes_result = bytes_a;
    double *result_gpu = new double[a.size()];

    hipMalloc(&d_values, bytes_values);
    hipMalloc(&d_columns, bytes_columns);
    hipMalloc(&d_indices, bytes_indices);
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_result, bytes_result);
    hipMemcpy(d_values, (const void *) matrix.values().data(), bytes_values, hipMemcpyHostToDevice);
    hipMemcpy(d_columns, (const void *) matrix.columns().data(), bytes_columns, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, (const void *) matrix.indices().data(), bytes_indices, hipMemcpyHostToDevice);
    hipMemcpy(d_a, (const void *) a.data(), bytes_a, hipMemcpyHostToDevice);

    hipEvent_t start_gpu, end_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&end_gpu);
    hipEventRecord(start_gpu);
    std::size_t block_size = 1024;
    std::size_t grid_size = (matrix.indices().size() + block_size - 1) / block_size;
    multiply<<<grid_size, block_size>>>(matrix.values().size(), matrix.indices().size(),
                                        d_values, d_columns, d_indices, d_a, d_result);
    hipEventRecord(end_gpu);
    hipEventSynchronize(end_gpu);
    float time_gpu = 0;
    hipEventElapsedTime(&time_gpu, start_gpu, end_gpu);
    hipEventDestroy(start_gpu);
    hipEventDestroy(end_gpu);

    hipMemcpy(result_gpu, d_result, bytes_a, hipMemcpyDeviceToHost);
    std::cerr << "GPU: " << "" << time_gpu << "ms" << std::endl;
    auto start_cpu = std::chrono::high_resolution_clock::now();
    auto result_cpu = matrix * a;
    auto end_cpu = std::chrono::high_resolution_clock::now();
    const float time_cpu = std::chrono::duration_cast<std::chrono::microseconds>(end_cpu - start_cpu).count() / 1000.0f;
    std::cerr << "CPU: " << time_cpu << "ms" << std::endl;

    hipFree(d_values);
    hipFree(d_columns);
    hipFree(d_indices);
    hipFree(d_a);
    hipFree(d_result);
    delete[] result_gpu;
    return 0;
}
