
#include <hip/hip_runtime.h>
#include <cstddef>
#include <iostream>
#include <chrono>
#include <vector>
#include <random>

__device__ void swap(double *a, double *b)
{
    if (a != b)
    {
        const double temp = *a;
        *a = *b;
        *b = temp;
    }
}

__global__ void bitonic_step(double *arr, std::size_t j, std::size_t k) {
    std::size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    std::size_t ixj = i ^ j;

    if (ixj > i) {
        if ((i & k) == 0) {
            if (arr[i] > arr[ixj])
                swap(arr + i, arr + ixj);
        } else {
            if (arr[i] < arr[ixj])
                swap(arr + i, arr + ixj);
        }
    }
}

bool is_sorted(const double *arr, const std::size_t n) {
    for (std::size_t i = 0; i < n - 1; i++)
        if (arr[i] > arr[i + 1])
            return false;
    return true;
}

void sort_cpu(std::vector<double> &arr) {
    for (int k = 2; k <= arr.size(); k *= 2) {
        for (int j = k/2; j > 0; j /= 2) {
            for (int i = 0; i < arr.size(); i++) {
                int ixj = i ^ j;
                if (ixj > i) {
                    if ((i & k) == 0) {
                        if (arr[i] > arr[ixj])
                            std::swap(arr[i], arr[ixj]);
                    } else {
                        if (arr[i] < arr[ixj])
                            std::swap(arr[i], arr[ixj]);
                    }
                }
            }
        }
    }
}

int main() {
    const std::size_t n = 128;

    std::mt19937 engine(112233);
    const int width = 100;
    std::uniform_int_distribution random_value(-width, width);
    std::vector<double> arr(n);
    for (double &value: arr)
        value = random_value(engine);

    double *d_arr;
    const std::size_t bytes_arr = arr.size() * sizeof(double);
    hipMalloc(&d_arr, bytes_arr);
    hipMemcpy(d_arr, (const void *) arr.data(), bytes_arr, hipMemcpyHostToDevice);
    double *result_gpu = new double[arr.size()];
    const std::size_t block_size = 256;
    const std::size_t grid_size = (n + block_size - 1) / block_size;
    hipEvent_t start_gpu, end_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&end_gpu);
    hipEventRecord(start_gpu);
    for (std::size_t k = 2; k <= n; k *= 2) {
        for (std::size_t j = k/2; j > 0; j /= 2) {
            bitonic_step<<<grid_size, block_size>>>(d_arr, j, k);
        }
    }
    hipEventRecord(end_gpu);
    hipEventSynchronize(end_gpu);
    float time_gpu;
    hipEventElapsedTime(&time_gpu, start_gpu, end_gpu);
    hipMemcpy(result_gpu, d_arr, bytes_arr, hipMemcpyDeviceToHost);
    std::cerr << "GPU: " << time_gpu << "ms, sorted = " << is_sorted(result_gpu, n) << std::endl;
    hipFree(d_arr);

    auto start_cpu = std::chrono::high_resolution_clock::now();
    sort_cpu(arr);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    const float time_cpu = std::chrono::duration_cast<std::chrono::microseconds>(end_cpu - start_cpu).count() / 1000.0f;
    std::cerr << "CPU: " << time_cpu << "ms, sorted = " << is_sorted(arr.data(), n) << std::endl;

    delete[] result_gpu;

    return 0;
}
